
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <hip/hip_fp16.h>
// #include <opencv2/opencv.hpp>
#include <stdexcept>
#include <vector>

#define cudaCheckError(status) { cudaAssert(status, __FILE__, __LINE__); }
inline auto cudaAssert(hipError_t status, const char* file, int line) -> void {
    if (status != hipSuccess) {
        fprintf(stderr, "CUDA API error \"%s\" at %s:%i\n", hipGetErrorString(status), file, line);
        exit(EXIT_FAILURE);
    }
}

#define cudnnCheckError(status) { cudnnAssert(status, __FILE__, __LINE__); }
inline auto cudnnAssert(cudnnStatus_t status, const char* file, int line) -> void {
    if (status != CUDNN_STATUS_SUCCESS) {
        fprintf(stderr, "CUDNN API error \"%s\" at %s:%i\n", cudnnGetErrorString(status), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CUDA_1D_KERNEL_LOOP(idx, n)                                 \
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < (n); \
       idx += blockDim.x * gridDim.x)

__global__
void float2half(const float* input, std::size_t size, half* output) {
    CUDA_1D_KERNEL_LOOP(idx, size) {
        output[idx] = __float2half(input[idx]);
    }
}

__global__
void half2float(const half* input, std::size_t size, float* output) {
    CUDA_1D_KERNEL_LOOP(idx, size) {
        output[idx] = __half2float(input[idx]);
    }
}


#define N 1 // Batches
#define C 1  // Channels
#define H 1024 // Hight
#define W H // Width


auto main(int argc, const char** argv) -> int {
    // if (argc != 2) {
    //     // std::cout << "usage: conv2d <filename>\n";
    //     printf("usage: conv2d <0: TENSOR_OP_MATH 1: DEFAULT_MATH>\n");
    //     return -1;
    // }

    // int op_type = atoi(argv[1]);
    // const char* filename = argv[1];
    // cv::Mat image = cv::imread(filename);
    // if (image.empty()) {
    //     throw std::runtime_error("cv::imread() failed: image not found");
    // }
    
    // image.convertTo(image, CV_32FC3);
    // cv::normalize(image, image, 0.0, 1.0, cv::NORM_MINMAX);

    // const std::size_t image_size = image.total() * image.channels();
    // const std::size_t image_size = N * C * H * W;
    const size_t image_size = N * C * H * W;
    
    cudnnHandle_t cudnn_handle;
    cudnnCheckError(cudnnCreate(&cudnn_handle));

    float* d_input = nullptr;
    cudaCheckError(hipMalloc(&d_input, image_size * sizeof(float)));
    // cudaCheckError(cudaMemcpy(d_input, image.ptr<float>(0), image_size * sizeof(float), cudaMemcpyDefault));
    
    half* fp16_image = nullptr;
    cudaCheckError(hipMalloc(&fp16_image, image_size * sizeof(half)));

    // float2half<<<1, 64>>>(d_input, image_size, fp16_image); // cpy
    hipFree(d_input);

    cudnnTensorDescriptor_t input_desc;
    cudnnCheckError(cudnnCreateTensorDescriptor(&input_desc));
    cudnnCheckError(cudnnSetTensor4dDescriptor(
        input_desc,
        CUDNN_TENSOR_NHWC,
        CUDNN_DATA_HALF,
        N, // 1,
        C, // image.channels(),
        H, // image.rows,
        W  // image.cols
    ));

    cudnnFilterDescriptor_t filter_desc;
    cudnnCheckError(cudnnCreateFilterDescriptor(&filter_desc));
    cudnnCheckError(cudnnSetFilter4dDescriptor(
        filter_desc,
        CUDNN_DATA_HALF,
        CUDNN_TENSOR_NCHW,
        C, // image.channels(),
        C, // image.channels(),
        3,
        3
    ));

    cudnnConvolutionDescriptor_t conv_desc;
    cudnnCheckError(cudnnCreateConvolutionDescriptor(&conv_desc));
    cudnnCheckError(cudnnSetConvolution2dDescriptor(
        conv_desc,
        1, 1,
        1, 1,
        1, 1,
        CUDNN_CROSS_CORRELATION,
        CUDNN_DATA_HALF
    ));

#if MODE == 0
    printf("Using CUDNN_DEFAULT_MATH.\n");
    cudnnCheckError(cudnnSetConvolutionMathType(conv_desc, CUDNN_DEFAULT_MATH));
#elif MODE == 1
    printf("Using CUDNN_TENSOR_OP_MATH.\n");
    cudnnCheckError(cudnnSetConvolutionMathType(conv_desc, CUDNN_TENSOR_OP_MATH));
#elif MODE == 2
    printf("Using CUDNN_DEFAULT_MATH.\n");
    cudnnCheckError(cudnnSetConvolutionMathType(conv_desc, CUDNN_DEFAULT_MATH));
#else
    printf("Using CUDNN_FMA_MATH.\n");
    cudnnCheckError(cudnnSetConvolutionMathType(conv_desc, CUDNN_FMA_MATH));
#endif

    cudnnTensorDescriptor_t output_desc;
    cudnnCheckError(cudnnCreateTensorDescriptor(&output_desc));
    cudnnCheckError(cudnnSetTensor4dDescriptor(
        output_desc,
        CUDNN_TENSOR_NHWC,
        CUDNN_DATA_HALF,
        1,
        C, // image.channels(),
        H, // image.rows,
        W  // image.cols
    ));

    cudnnConvolutionFwdAlgo_t fwd_algo;
    int requested_algo_count;
    int algo_count;

    cudnnCheckError(cudnnGetConvolutionForwardAlgorithmMaxCount(cudnn_handle, &requested_algo_count));
    std::vector<cudnnConvolutionFwdAlgoPerf_t> perf_results(requested_algo_count);
    cudnnCheckError(cudnnFindConvolutionForwardAlgorithm(
        cudnn_handle,
        input_desc,
        filter_desc,
        conv_desc,
        output_desc,
        requested_algo_count,
        &algo_count,
        perf_results.data()
    ));

    fwd_algo = perf_results.front().algo;

    std::size_t workspace_size = 0;
    cudnnCheckError(cudnnGetConvolutionForwardWorkspaceSize(
        cudnn_handle,
        input_desc,
        filter_desc,
        conv_desc,
        output_desc,
        fwd_algo,
        &workspace_size
    ));

    void* d_workspace = nullptr;
    cudaCheckError(hipMalloc(&d_workspace, workspace_size));

    const std::vector<half> filter = {
        0.0625, 0.125, 0.0625,
        0.125, 0.25, 0.125,
        0.0625, 0.125, 0.0625
    };

    std::vector<half> h_filter;
    for (std::size_t idx = 0; idx < 3 * 3; ++idx) {
        for (const auto& val : filter) {
            h_filter.emplace_back(val);
        }
    }
    
    half* d_filter = nullptr;
    const std::size_t filter_size = h_filter.size();
    cudaCheckError(hipMalloc(&d_filter, filter_size * sizeof(half)));
    cudaCheckError(hipMemcpy(d_filter, h_filter.data(), filter_size * sizeof(half), hipMemcpyDefault));
    
    half* d_output = nullptr;
    cudaCheckError(hipMalloc(&d_output, image_size * sizeof(half)));

    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;

    cudnnCheckError(cudnnConvolutionForward(
        cudnn_handle,
        &alpha,
        input_desc,
        fp16_image,
        filter_desc,
        d_filter,
        conv_desc,
        fwd_algo,
        d_workspace,
        workspace_size,
        &beta,
        output_desc,
        d_output
    ));

    float* fp32_output = nullptr;
    cudaCheckError(hipMalloc(&fp32_output, image_size * sizeof(float)));

    half2float<<<1, 64>>>(d_output, image_size, fp32_output);
    hipFree(d_output);
    
    // cv::Mat output(image.rows, image.cols, CV_32FC3);
    // cudaCheckError(cudaMemcpy(output.ptr<float>(0), fp32_output, image_size * sizeof(float), cudaMemcpyDefault));

    // cv::normalize(output, output, 0.0, 255.0, cv::NORM_MINMAX);
    // output.convertTo(output, CV_8UC3);

    // cv::imshow("output", output);
    // cv::waitKey();

    // cv::imwrite("output.png", output);

    hipFree(d_filter);
    hipFree(d_workspace);
    hipFree(fp16_image);
    hipFree(fp32_output);

    cudnnDestroyTensorDescriptor(input_desc);
    cudnnDestroyFilterDescriptor(filter_desc);
    cudnnDestroyConvolutionDescriptor(conv_desc);
    cudnnDestroyTensorDescriptor(output_desc);

    cudnnDestroy(cudnn_handle);
    printf("Conv Done.\n");
    return 0;
}
